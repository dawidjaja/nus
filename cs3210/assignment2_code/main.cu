#include "hip/hip_runtime.h"
#include <cinttypes>
#include <chrono>
#include <iostream>
#include <limits>
#include <sstream>
#include "hash.h"

#define FREEZE_TIMESTAMP 0

// Big endian. Please reverse the bit position labels in the pdf next time.
union Input {
    struct {
        uint32_t dummy; // for alignment
        uint8_t prev_digest[32];
        uint32_t timestamp;
        uint64_t txn_id;
        uint64_t nonce;
    } as_parts;
    struct {
        uint32_t dummy; // for alignment
        uint8_t bytes[52];
    } as_bytes;
};

__managed__ Input input;
__managed__ uint8_t digest[32];
__managed__ bool is_done;
__managed__ uint64_t n;

uint64_t ascii_to_u64(const std::string str) {
    uint64_t result;
    std::istringstream iss(str);

    for (size_t i = 0; i < sizeof(uint64_t); i++) {
        uint8_t byte;
        iss >> byte;
        result = result << 8 | byte;
    }

    return result;
}

void check_cuda_errors() {
    hipError_t rc;
    rc = hipGetLastError();
    if (rc != hipSuccess)
    {
        std::cout << "Last CUDA error: " << hipGetErrorString(rc) << std::endl;
    }
}

__global__ void work() {
    Input localInput = input;
    uint8_t res[32];

    uint64_t start;
    size_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    size_t numThreads = gridDim.x * blockDim.x;
    start = (threadId * (ULLONG_MAX / numThreads));

    for (uint64_t nonce = start;; nonce++) {
        if (is_done) {
            break;
        }

        localInput.as_parts.nonce = nonce;
        sha256(res, localInput.as_bytes.bytes, 52);

        uint64_t prefix = 0;
        for (size_t i = 0; i < 8; i++) {
            prefix = (prefix << 8) | res[i];
        }
        if (prefix < n) {
            atomicExch((unsigned long long *)&input.as_parts.nonce, (unsigned long long)nonce);
            is_done = 1;
        }
        if (nonce == ULLONG_MAX) {
            break;
        }
    }
}

__global__ void get_final_digest() {
    sha256(digest, input.as_bytes.bytes, 52);
}

int main(int argc, char **argv) {
    srand(0);

    std::string prev_digest;
    std::cin >> prev_digest;
    const char *prev_digest_chars = prev_digest.c_str();
    for (size_t i = 0; i < 32 * sizeof(uint8_t); i++) {
        sscanf(prev_digest_chars + 2 * i, "%02hhx", input.as_parts.prev_digest + i);
    }

    // TODO: Generate this
#if FREEZE_TIMESTAMP
    uint32_t timestamp = 1601555562;
#else
    uint32_t timestamp = std::chrono::duration_cast<std::chrono::seconds>(
        std::chrono::system_clock::now().time_since_epoch()
    ).count();
#endif
    std::string raw_tid;
    std::cin >> raw_tid;
    uint64_t tid = ascii_to_u64(raw_tid);

    // uint64_t n;
    std::cin >> n;
    // TODO: Remove this
    // n = 1;
    // n <<= 60;

    input.as_parts.timestamp = __builtin_bswap32(timestamp);
    input.as_parts.txn_id = __builtin_bswap64(tid);

    int BLOCKS, THREADS;

    sscanf(argv[1], "%d", &BLOCKS);
    sscanf(argv[2], "%d", &THREADS);

    work<<<BLOCKS, THREADS>>>();
    hipDeviceSynchronize();
    check_cuda_errors();

    get_final_digest<<<1, 1>>>();
    hipDeviceSynchronize();
    check_cuda_errors();

    std::cout << __builtin_bswap32(input.as_parts.timestamp) << std::endl;
    std::cout << __builtin_bswap64(input.as_parts.nonce) << std::endl;
    for (size_t i = 0; i < 32; i++) {
        printf("%02x", digest[i]);
    }
    printf("\n");
}
