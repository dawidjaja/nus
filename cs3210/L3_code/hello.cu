/*
 * Hello World in CUDA
 *
 * CS3210
 *
 * This program start from "hello world" string and should print "HELLO WORLD"
 *
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N       32 

// #define      DISCRETE

__global__ void hello(char *a, int len)
{
        int tid = blockIdx.z * 32 + blockIdx.y * 16 + blockIdx.x * 8 + threadIdx.x * 4 + threadIdx.y;
        if (tid >= len)
                return;
        a[tid] += 'A' - 'a';
}

int main()
{
        // original string
        char a[N] = "hello@world";
        // length
        int len = strlen(a);
        // pointer to the string on device
        char* ad;
        // pointer to the final string on host
        char* ah;
        // CUDA returned error code
        hipError_t rc;


        //allocate space for the string on device (GPU) memory
        hipMalloc((void**)&ad, N);
        hipMemcpy(ad, a, N, hipMemcpyHostToDevice);

        dim3 gridDimensions (2, 2, 2);
        dim3 blockDimensions (2, 4);

        // launch the kernel
        hello<<<gridDimensions, blockDimensions>>>(ad, len);
        hipDeviceSynchronize();

	// for discrete GPUs, get the data from device memory to host memory
        hipMemcpy(a, ad, N, hipMemcpyDeviceToHost);
        ah = a;

        // was there any error?
        rc = hipGetLastError();
        if (rc != hipSuccess)
                printf("Last CUDA error %s\n", hipGetErrorString(rc));

        // print final string
        printf("%s!\n", ah);

        // free memory
        hipFree(ad);

        return 0;
}

